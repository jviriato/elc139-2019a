#include "hip/hip_runtime.h"
#include <complex>
#include <iostream>
#include <sys/time.h>
#include <thrust/complex.h>

using namespace std;

__global__ void make_fractal(char *mat, int max_n, int max_row, int max_column){

   int r = threadIdx.x;

   using namespace std;


		for(int c = 0; c < max_column; ++c){
			thrust::complex<float> z;
			int n = 0;

         while(z.real() < 2 && ++n < max_n)
				z = pow(z, 2) + decltype(z)(
					(float)c * 2 / max_column - 1.5,
					(float)r * 2 / max_row - 1
				);
			mat[r*max_column + c]=(n == max_n ? '#' : '.');
		}


}

int main(int argc, char *argv[]){
	int max_row, max_column, max_n, print;
	cin >> max_row;
	cin >> max_column;
	cin >> max_n;
	print = 0;
	char *mat;
	hipMallocManaged(&mat, max_row*max_column*sizeof(unsigned char));





   timeval start, end;
   gettimeofday(&start, NULL);

   make_fractal<<<1, max_row>>>(mat, max_n, max_row, max_column);

   hipDeviceSynchronize();


	gettimeofday(&end, NULL);
   double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
   std::cout << "compute time: " << runtime << " s\n";

   if(print){
      for(int r = 0; r < max_row; ++r){
         for(int c = 0; c < max_column; ++c)
            std::cout << mat[r*max_column + c];
         cout << '\n';
      }
   }
}

