#include "hip/hip_runtime.h"
#include <cstdlib>
#include <sys/time.h>
#include <math.h>
#include "wave.h"

__global__
void calculateFrames(unsigned char* pic, int width, int frames)
{
  int i = threadIdx.x; 
  int offset = blockDim.x; 
  for (int frame = i; frame < frames; frame += offset) {
    for (int row = 0; row < width; row++) {
      for (int col = 0; col < width; col++) {
        float fx = col - 1024/2;
        float fy = row - 1024/2;
        float d = sqrtf( fx * fx + fy * fy );
        unsigned char color = (unsigned char) (160.0f + 127.0f *
                                          cos(d/10.0f - frame/7.0f) /
                                          (d/50.0f + 1.0f));

        pic[frame * width * width + row * width + col] = (unsigned char) color;
      }
    }
  }
}


int main(int argc, char *argv[])
{

    // check command line
    if (argc != 3) {fprintf(stderr, "usage: %s frame_width num_frames\n", argv[0]); exit(-1);}
    int width = atoi(argv[1]);
    if (width < 100) {fprintf(stderr, "error: frame_width must be at least 100\n"); exit(-1);}
    int frames = atoi(argv[2]);
    if (frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
    printf("computing %d frames of %d by %d picture\n", frames, width, width);
    
     // allocate picture array
    unsigned char* pic;
    hipMallocManaged(&pic, frames * width * width * sizeof(unsigned char));

    timeval start, end;
    gettimeofday(&start, NULL);

    calculateFrames<<<1,frames>>>(pic,width, frames);
    

    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
    printf("%.4f\n", runtime);
    // verify result by writing frames to BMP files
    if ((frames <= 100)) {
      for (int frame = 0; frame < frames; frame++) {
        char name[32];
        sprintf(name, "wave%d.bmp", frame + 1000);
        writeBMP(width, width, &pic[frame * width * width], name);
      }
    }

    hipFree(pic);
    return 0;

}

